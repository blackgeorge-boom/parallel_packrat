#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */ 
#include <stdio.h>
#include "dmv.h"

/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */ 
__device__ int get_global_tid()
{
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */ 
__device__ int get_local_tid()
{
    return blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Naive kernel
 */ 
__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    /*
     * FILLME: fill the code for the naive kernel.
     */ 
}

/*
 *  Coalesced memory acceses
 */
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
    /*
     * FILLME: fill the code for the coalesced kernel.
     */ 
}

/*
 *  Use of shared memory
 */
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    /*
     * FILLME: fill the code for the shared memory kernel.
     */ 
}
