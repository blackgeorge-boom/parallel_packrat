/*
 *  dmv_main.cu -- DMV front-end program.
 *
 *  Copyright (C) 2010-2012, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2012, Vasileios Karakasis
 */ 

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "alloc.h"
#include "error.h"
#include "timer.h"
#include "dmv.h"
#include "gpu_util.h"

#ifndef VALUES_MAX
#   define VALUES_MAX MAKE_VALUE_CONSTANT(1.)
#endif

#ifndef EPS
#   define EPS MAKE_VALUE_CONSTANT(1.e-6)
#endif

#ifndef NR_ITER
#   define NR_ITER 100
#endif

static void check_result(const value_t *test, const value_t *orig, size_t n)
{
    printf("Checking ... ");
    size_t  i_fail = vec_equals(test, orig, n, EPS);
    if (!i_fail) {
        printf("PASSED\n");
    } else {
        printf("FAILED (index: %ld)\n", i_fail - 1);
        printf("%" VALUE_FORMAT " != " "%" VALUE_FORMAT "\n",
               test[i_fail-1], orig[i_fail-1]);
    }
}

static void report_results(xtimer_t *timer, size_t n)
{
    double  elapsed_time = timer_elapsed_time(timer);
    size_t  flops        = 2*n*n*NR_ITER;

    printf("Elapsed time: %lf s\n", elapsed_time);
    printf("Performance:  %lf Gflop/s\n", flops*1.e-9 / elapsed_time);
}

static void print_usage()
{
    printf("Usage: [GPU_KERNEL=<kernel_no>] [GPU_BLOCK_SIZE=<size>] "
           "%s <matrix size>\n", program_name);
    printf("GPU_KERNEL defaults to 0\n");
    printf("GPU_BLOCK_SIZE defaults to 256\n");
    printf("Available kernels [id:name]:\n");
    size_t i;
    for (i = 0; i < GPU_KERNEL_END; ++i) {
        printf("\t%zd:%s\n", i, gpu_kernels[i].name);
    }
}

int main(int argc, char **argv)
{
    set_program_name(argv[0]);
    if (argc < 2) {
        warning(0, "too few arguments");
        print_usage();
        exit(EXIT_FAILURE);
    }

    size_t n = atoi(argv[1]);
    if (!n)
        error(0, "invalid argument: %s", argv[1]);

    /* Read block size and kernel to launch from the environment */
    const char *env_gpu_kernel = getenv("GPU_KERNEL");
    const char *env_gpu_block_size = getenv("GPU_BLOCK_SIZE");
    int kernel = (env_gpu_kernel) ? atoi(env_gpu_kernel) : GPU_NAIVE;
    int block_size = (env_gpu_block_size) ? atoi(env_gpu_block_size) : 256;
    size_t orig_n = n;  // original matrix size
    int grid_size = 1;  // FILLME: compute the grid size

    /*
     *  FILLME: you can optionally adjust appropriately (increase
     *          only) the matrix size here if that helps you with your
     *          kernel code, e.g., to avoid divergent warps.
     */ 

    printf("Matrix size: %zd\n", orig_n);
    printf("Adjusted matrix size: %zd\n", n);

    /*
     * Allocate the structures.
     * 
     * Initialization to zero is crucial if you adjusted the matrix
     * size.
     */
    value_t **A = (value_t **) calloc_2d(n, n, sizeof(**A));
    if (!A)
        error(1, "alloc_2d failed");

    value_t *x = (value_t *) calloc(n, sizeof(*x));
    if (!x)
        error(1, "malloc failed");

    value_t *y_serial = (value_t *) calloc(n, sizeof(*y_serial));
    if (!y_serial)
        error(1, "malloc failed");
    
    value_t *y = (value_t *) calloc(n, sizeof(*y));
    if (!y)
        error(1, "malloc failed");

    /* Initialize */
    srand48(0);
    mat_init_rand(A, orig_n, VALUES_MAX);
    vec_init_rand(x, orig_n, VALUES_MAX);
    vec_init(y_serial, orig_n, MAKE_VALUE_CONSTANT(0.0));
    vec_init(y, orig_n, MAKE_VALUE_CONSTANT(0.0));

    /* Setup timers */
    xtimer_t timer;

    /* Compute serial */
#ifdef SERIAL_KERNEL
    printf(">>>> Begin of record <<<<\n");
    printf("Serial version:\n");
    timer_clear(&timer);
    timer_start(&timer);
    for (size_t i = 0; i < NR_ITER; ++i)
        dmv_serial(A, x, y_serial, orig_n);
    timer_stop(&timer);
    report_results(&timer, orig_n);
    printf(">>>> End of record <<<<\n");
#endif  // SERIAL_KERNEL

#ifdef OPENMP_KERNEL
    /* Compute OpenMP */
    printf(">>>> Begin of record <<<<\n");
    printf("OpenMP version:\n");
    timer_clear(&timer);
    timer_start(&timer);
    for (size_t i = 0; i < NR_ITER; ++i)
        dmv_omp(A, x, y, orig_n);
    timer_stop(&timer);
#ifndef _NOCHECK_
    check_result(y, y_serial, orig_n);
#endif
    report_results(&timer, orig_n);
    printf(">>>> End of record <<<<\n");
#endif  // OPENMP_KERNEL

#ifdef GPU_KERNEL
    /*
     *  FILLME: Set up the blocks, grid and shared memory depending on
     *          the kernel. Make any transformations to the input
     *          matrix here.
     */ 

    dim3 gpu_block(1, 1);   // FILLME: set up the block dimensions
    dim3 gpu_grid(1, 1);    // FILLME: set up the grid dimensions
    size_t shmem_size = 0;  // FILLME: set up the shared memory size

    printf(">>>> Begin of record <<<<\n");
    printf("Block size: %dx%d\n", gpu_block.x, gpu_block.y);
    printf("Grid size : %dx%d\n", gpu_grid.x, gpu_grid.y);
    printf("Shared memory size: %ld bytes\n", shmem_size);

    /* GPU allocations */
    value_t *gpu_A = (value_t *) gpu_alloc(n*n*sizeof(*gpu_A));
    if (!gpu_A)
        error(0, "gpu_alloc failed: %s", gpu_get_last_errmsg());
    
    value_t *gpu_x = (value_t *) gpu_alloc(n*sizeof(*gpu_x));
    if (!gpu_x)
        error(0, "gpu_alloc failed: %s", gpu_get_last_errmsg());

    value_t *gpu_y = (value_t *) gpu_alloc(n*sizeof(*gpu_y));
    if (!gpu_y)
        error(0, "gpu_alloc failed: %s", gpu_get_last_errmsg());
    
    /* Copy data to GPU */
    if (copy_to_gpu(A[0], gpu_A, n*n*sizeof(*gpu_A)) < 0)
        error(0, "copy_to_gpu failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(x, gpu_x, n*sizeof(*gpu_x)) < 0)
        error(0, "copy_to_gpu failed: %s", gpu_get_last_errmsg());

    /* Reset y and copy it to GPU */
    vec_init(y, n, MAKE_VALUE_CONSTANT(0.0));
    if (copy_to_gpu(y, gpu_y, n*sizeof(*gpu_y)) < 0)
        error(0, "copy_to_gpu failed: %s", gpu_get_last_errmsg());

    if (kernel >= GPU_KERNEL_END)
        error(0, "the requested kernel does not exist");

    printf("GPU kernel version: %s\n", gpu_kernels[kernel].name);

    /* Execute and time the kernel */
    timer_clear(&timer);
    timer_start(&timer);
    for (size_t i = 0; i < NR_ITER; ++i) {
        gpu_kernels[kernel].fn<<<gpu_grid,gpu_block,shmem_size>>>
            (gpu_A, gpu_x, gpu_y, n);
#ifdef _DEBUG_
        hipError_t err;
        if ( (err = hipGetLastError()) != hipSuccess)
            error(0, "gpu kernel failed to launch: %s", gpu_get_errmsg(err));
#endif
        hipDeviceSynchronize();
    }
    timer_stop(&timer);

    /* Copy result back to host and check */
    if (copy_from_gpu(y, gpu_y, n*sizeof(*y)) < 0)
        error(0, "copy_from_gpu failed: %s", gpu_get_last_errmsg());

#ifndef _NOCHECK_
    check_result(y, y_serial, orig_n);
#endif
    report_results(&timer, orig_n);
    printf(">>>> End of record <<<<\n");
#endif  // GPU_KERNEL 

    /* Free resources on host */
    free_2d((void **) A);
    free(x);
    free(y);
    free(y_serial);

#ifdef GPU_KERNEL
    /* Free resources on GPU */
    gpu_free(gpu_A);
    gpu_free(gpu_x);
    gpu_free(gpu_y);
#endif  // GPU_KERNEL 

    return EXIT_SUCCESS;
}
